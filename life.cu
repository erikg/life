#define CUDA 1
extern "C" {

#include "life.inl"

void life_init(int width_, int height_) {
	int bytes = (width_ + 2) * (height_ + 2) * sizeof(cell_t);
	width = width_;
	height = height_;
	hipMalloc(tile, bytes);
	hipMalloc(tile+1, bytes);
	local_tile = (cell_t *)malloc(bytes);
}

void life_sim() {
	life_sim_cell<<<dim3(WIDTH,HEIGHT),1>>>(tile[currBuffer], tile[1-currBuffer], width);

	/*
	for(int i=0;i<height+2;i++) {
		cell_t *dst = tile[currBuffer]+ (i * width * sizeof(cell_t));
		dst[0].val = dst[width].val = 0;
	}
	*/
	currBuffer = 1-currBuffer;
}


void life_deinit() {
	hipFree(tile[0]);
	hipFree(tile[1]);
	tile[0] = tile[1] = 0;
	width = height = 0;
}

void life_load(cell_t *buf, int w, int h, int off_x, int off_y) {
	int j;
	for(j=1;j<h;j++) {
		hipMemcpy(tile[currBuffer] + ((off_y+j+1)*width) + off_x + 1, buf+(j*w), w, hipMemcpyHostToDevice);
	}
}

cell_t *life_buffer() {
	hipMemcpy(local_tile, tile[currBuffer], width*height*sizeof(cell_t), hipMemcpyDeviceToHost);
	return local_tile;
}

}
