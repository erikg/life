#define CUDA 1
extern "C" {

#include "life.inl"

void life_init(int width_, int height_) {
	int bytes = (width_ + 2) * (height_ + 2) * sizeof(cell_t);
	width = width_;
	height = height_;
	hipMalloc(tile, bytes);
	hipMalloc(tile+1, bytes);
	local_tile = (cell_t *)malloc(bytes);
}

void life_deinit() {
	hipFree(tile[0]);
	hipFree(tile[1]);
	tile[0] = tile[1] = 0;
	width = height = 0;
}

void life_sim() {
	life_sim_cell<<<dim3(WIDTH-2,HEIGHT-2),1>>>(tile[currBuffer], tile[1-currBuffer], width);
	hipMemset(tile[currBuffer], 0, (WIDTH+2)*(HEIGHT+2)*sizeof(cell_t));
	currBuffer = 1-currBuffer;
}

void life_load(cell_t *buf, int w, int h, int off_x, int off_y) {
	int j;
	for(j=1;j<h;j++) {
		hipMemcpy(tile[currBuffer] + ((off_y+j+1)*width) + off_x + 1, buf+(j*w), w, hipMemcpyHostToDevice);
	}
}

cell_t *life_buffer() {
	hipMemcpy(local_tile, tile[currBuffer], width*height*sizeof(cell_t), hipMemcpyDeviceToHost);
	return local_tile;
}

}
